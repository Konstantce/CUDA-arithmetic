#include <stdio.h>
#include <hip/hip_runtime.h>

int main(int argc, char **argv)
{
    hipDeviceProp_t dP;

    int rc = hipGetDeviceProperties(&dP, 0);
    if(rc != hipSuccess)
    {
        hipError_t error = hipGetLastError();
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return rc; /* Failure */
    }
    else
    {
        printf("%d%d", dP.major, dP.minor);
        return 0; /* Success */
    }
}