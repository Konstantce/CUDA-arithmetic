#include "hip/hip_runtime.h"
#include "cuda_structs.h"

//FFT (we propose very naive realization)
//----------------------------------------------------------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------------------------------------

//Sources of inspiration:
//http://www.staff.science.uu.nl/~bisse101/Articles/preprint1138.pdf
//https://cs.wmich.edu/gupta/teaching/cs5260/5260Sp15web/studentProjects/tiba&hussein/03278999.pdf
//http://users.umiacs.umd.edu/~ramani/cmsc828e_gpusci/DeSpain_FFT_Presentation.pdf
//http://www.bealto.com/gpu-fft_intro.html
//https://github.com/mmajko/FFT-cuda/blob/master/src/fft-cuda.cu
//Also have a loot at GPU gems

//NB: arr should be a power of two


//commom FFT routines
//------------------------------------------------------------------------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------------------------------------------------

struct field_pair
{
	embedded_field a;
	embedded_field b;
};

DEVICE_FUNC field_pair __inline__ fft_buttefly(const embedded_field& x, const embedded_field& y, const embedded_field& root_of_unity)
{
	embedded_field temp = y * root_of_unity;
	return field_pair{ x + temp, x - temp};
}

DEVICE_FUNC embedded_field __inline__ get_root_of_unity(uint32_t index, uint32_t omega_idx_coeff = 1, bool inverse = false)
{
	embedded_field result(EMBEDDED_FIELD_R);
	uint32_t real_idx = index * omega_idx_coeff;
	if (inverse)
		real_idx = (1 << ROOTS_OF_UNTY_ARR_LEN) - real_idx;
	for (unsigned k = 0; k < ROOTS_OF_UNTY_ARR_LEN; k++)
	{
		if (CHECK_BIT(real_idx, k))
			result *= embedded_field(EMBEDDED_FIELD_ROOTS_OF_UNITY[k]);
	}
	return result;	
}

struct geometry
{
    int gridSize;
    int blockSize;
};

template<typename T>
geometry find_suitable_geometry(T func, uint shared_memory_used, uint32_t smCount)
{
    int gridSize;
    int blockSize;
    int maxActiveBlocks;

    hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, func, shared_memory_used, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, func, blockSize, shared_memory_used);
    gridSize = maxActiveBlocks * smCount;

    return geometry{gridSize, blockSize};
}

//Naive FFT-realization
//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------

__global__ void FFT_shuffle(embedded_field* __restrict__ input_arr, embedded_field* __restrict__ output_arr, uint32_t arr_len, uint32_t log_arr_len)
{
	uint32_t  tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < arr_len)
	{
		output_arr[tid] = input_arr[__brev(tid) >> (32 - log_arr_len)];
		tid += blockDim.x * gridDim.x;
	}
}

__global__ void FFT_iteration(embedded_field* __restrict__ input_arr, embedded_field* __restrict__ output_arr, 
	uint32_t arr_len, uint32_t log_arr_len, uint32_t step)
{
	uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t k = (1 << step);
	uint32_t l = 2 * k;
	size_t omega_coeff = 1 << (ROOTS_OF_UNTY_ARR_LEN - log_arr_len);
	while (i < arr_len / 2)
	{
		uint32_t first_index = l * (i / k) + (i % k);
		uint32_t second_index = first_index + k;

		uint32_t root_of_unity_index = (1 << (log_arr_len - step - 1)) * (i % k); 
		embedded_field omega = get_root_of_unity(root_of_unity_index, omega_coeff);

		field_pair ops = fft_buttefly(input_arr[first_index], input_arr[second_index], omega);

		output_arr[first_index] = ops.a;
		output_arr[second_index] = ops.b;

		i += blockDim.x * gridDim.x;
	}
}

#include <iostream>

void naive_fft_driver(embedded_field* input_arr, embedded_field* output_arr, uint32_t arr_len, bool is_inverse_FFT = false)
{
	//first check that arr_len is a power of 2

	uint log_arr_len = BITS_PER_LIMB - __builtin_clz(arr_len) - 1;
	std::cout << "Log arr len: " << log_arr_len << std::endl;
    assert(arr_len = (1 << log_arr_len));

	//find optimal geometry

	hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
	uint32_t smCount = prop.multiProcessorCount;

	geometry FFT_shuffle_geometry = find_suitable_geometry(FFT_shuffle, 0, smCount);
	geometry FFT_iter_geometry = find_suitable_geometry(FFT_iteration, 0, smCount);

	//allocate additional memory

	embedded_field* additional_device_memory = nullptr;
	hipError_t cudaStatus = hipMalloc((void **)&additional_device_memory, arr_len * sizeof(embedded_field));
	
	//FFT shuffle;

	embedded_field* temp_output_arr = (log_arr_len % 2 ? additional_device_memory : output_arr);
	embedded_field* temp_input_arr = (log_arr_len % 2 ? output_arr : additional_device_memory);
	FFT_shuffle<<<FFT_shuffle_geometry.gridSize, FFT_shuffle_geometry.blockSize>>>(input_arr, temp_output_arr, arr_len, log_arr_len);
	
	//FFT main cycle

	for (uint32_t step = 0; step < log_arr_len; step++)
	{
		//swap input and iutput arrs

		embedded_field* swap_arr = temp_input_arr;
		temp_input_arr = temp_output_arr;
		temp_output_arr = swap_arr;
		
		FFT_iteration<<<FFT_iter_geometry.gridSize, FFT_iter_geometry.blockSize>>>(temp_input_arr, temp_output_arr, arr_len, log_arr_len, step);
	}

	//clean_up
	hipFree(additional_device_memory);
}


//Bellman FFT-realization
//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------

//TODO: make the same things using shuffle instructions and shared memory

DEVICE_FUNC void _basic_serial_radix2_FFT(embedded_field* arr, size_t log_arr_len, size_t omega_idx_coeff, bool is_inverse_FFT)
{
	size_t tid = threadIdx.x;
	size_t arr_len = 1 << log_arr_len;

	for(size_t i = tid; i < arr_len; i+= blockDim.x)
	{	
		size_t rk = __brev(i) >> (32 - log_arr_len);
		if (i < rk)
		{	
			embedded_field temp = arr[i];
			arr[i] = arr[rk];
			arr[rk] = temp;
		}
	}

	__syncthreads();
	
    for (size_t step = 0; step < log_arr_len; ++step)
    {
        uint32_t i = tid;
		uint32_t k = (1 << step);
		uint32_t l = 2 * k;
		while (i < arr_len / 2)
		{
			uint32_t first_index = l * (i / k) + (i % k);
			uint32_t second_index = first_index + k;

			uint32_t omega_idx = (1 << (log_arr_len - step - 1)) * (i % k); 
			embedded_field omega = get_root_of_unity(omega_idx, omega_idx_coeff, is_inverse_FFT);

			field_pair ops = fft_buttefly(arr[first_index], arr[second_index], omega);

			arr[first_index] = ops.a;
			arr[second_index] = ops.b;

			i += blockDim.x;
		}
		
		__syncthreads();
	}
}

__global__ void _basic_parallel_radix2_FFT(const embedded_field* input_arr, embedded_field* output_arr, embedded_field* temp_arr_base, 
	size_t log_arr_len, size_t log_num_subblocks, bool is_inverse_FFT)
{
	assert( log_arr_len <= ROOTS_OF_UNTY_ARR_LEN && "the size of array is too large for FFT");

	size_t omega_coeff = 1 << (ROOTS_OF_UNTY_ARR_LEN - log_arr_len);
	size_t L = 1 << (log_arr_len - log_num_subblocks);
	size_t NUM_SUBBLOCKS = 1 << log_num_subblocks;

	embedded_field* temp_arr = temp_arr_base + L * blockIdx.x;

	embedded_field omega_step = get_root_of_unity(blockIdx.x * L, omega_coeff, is_inverse_FFT);
        
    for (size_t i = threadIdx.x; i < L; i+= blockDim.x)
    {
        embedded_field omega_init = get_root_of_unity(blockIdx.x * i, omega_coeff, is_inverse_FFT);
		temp_arr[i] = embedded_field::zero();
		for (size_t s = 0; s < NUM_SUBBLOCKS; ++s)
        {
            size_t idx = i + s * L;
            temp_arr[i] += input_arr[idx] * omega_init;
            omega_init *= omega_step;
        }
	}

	__syncthreads();

	_basic_serial_radix2_FFT(temp_arr, log_arr_len - log_num_subblocks, NUM_SUBBLOCKS * omega_coeff, is_inverse_FFT);

	for (size_t i = threadIdx.x; i < L; i+= blockDim.x)
		output_arr[i * NUM_SUBBLOCKS + blockIdx.x] = temp_arr[i];
}

__global__ void _radix2_one_block_FFT(const embedded_field* input_arr, embedded_field* output_arr, size_t log_arr_len, bool is_inverse_FFT)
{
	extern __shared__ embedded_field temp_arr[];
	size_t arr_len = 1 << log_arr_len;
	size_t omega_coeff = 1 << (ROOTS_OF_UNTY_ARR_LEN - log_arr_len);

	
	for (size_t i = threadIdx.x; i < arr_len; i+= blockDim.x)
	{
		temp_arr[i] = input_arr[i];
	}

	_basic_serial_radix2_FFT(temp_arr, log_arr_len, omega_coeff, is_inverse_FFT);

	for (size_t i = threadIdx.x; i < arr_len; i+= blockDim.x)
		output_arr[i] = temp_arr[i];
}

geometry find_geometry_for_advanced_FFT(uint arr_len)
{
	//TODO: this particular values are customized for my architecture

	size_t DEFAULT_FFT_GRID_SIZE = 8;
	size_t DEFAULT_FFT_BLOCK_SIZE = 512;

	geometry res;

	if (arr_len  <  2 * DEFAULT_FFT_BLOCK_SIZE)
	{
		res.gridSize = 1;
		res.blockSize = max(arr_len / 2, 1);
	}
	else
	{
		res.gridSize = min(DEFAULT_FFT_GRID_SIZE, arr_len / (2 * DEFAULT_FFT_BLOCK_SIZE));
		res.blockSize = min(DEFAULT_FFT_BLOCK_SIZE, (size_t)(arr_len / (2 * res.gridSize)));
	}
	
	std::cout << "grid_size: " << res.gridSize << ", block size: " << res.blockSize << std::endl;
	return res;
}

void advanced_fft_driver(embedded_field* input_arr, embedded_field* output_arr, uint32_t arr_len, bool is_inverse_FFT = false)
{
	//first check that arr_len is a power of 2

	uint log_arr_len = BITS_PER_LIMB - __builtin_clz(arr_len) - 1;
    assert(arr_len = (1 << log_arr_len));

	geometry kernel_geometry = find_geometry_for_advanced_FFT(arr_len);

	if (kernel_geometry.gridSize == 1)
	{
		std::cout << "1block FFT - serial" << std::endl;
		
		_radix2_one_block_FFT<<<1, kernel_geometry.blockSize, kernel_geometry.blockSize * 2 * sizeof(embedded_field)>>>(input_arr, output_arr, 
			log_arr_len, is_inverse_FFT);
		hipDeviceSynchronize();

		return;
	}

	size_t num_of_blocks = kernel_geometry.gridSize;
	uint log_num_subblocks = BITS_PER_LIMB - __builtin_clz(num_of_blocks) - 1;
	size_t block_size = 1 << (log_arr_len - log_num_subblocks);

	//allocate temporary memory
	embedded_field* temp_memory = nullptr;
	hipError_t cudaStatus = hipMalloc((void **)&temp_memory, num_of_blocks * block_size * sizeof(embedded_field));

	_basic_parallel_radix2_FFT<<<kernel_geometry.gridSize, kernel_geometry.blockSize>>>(input_arr, output_arr, temp_memory,
		log_arr_len, log_num_subblocks, is_inverse_FFT);
	hipDeviceSynchronize();

	hipFree(temp_memory);
}

#define FFT_DRIVER(input_arr, output_arr, arr_len, is_inverse_FFT) advanced_fft_driver(input_arr, output_arr, arr_len, is_inverse_FFT)


//polynomial multiplication via FFT

struct polynomial
{
	size_t deg;
	embedded_field* coeffs;
};

size_t get_power_of_two(size_t n)
{
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;

    return n;
}

__global__ void _mul_vecs(const embedded_field* a_arr, const embedded_field* b_arr, embedded_field* c_arr, size_t arr_len)
{
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < arr_len)
	{
		c_arr[tid] = (a_arr[tid] * b_arr[tid]);
		tid += blockDim.x * gridDim.x;
	}
}

void _mul_vecs_driver(const embedded_field* a_arr, const embedded_field* b_arr, embedded_field* c_arr, size_t arr_len)
{
	int blockSize;
  	int minGridSize;
  	int realGridSize;
	int maxActiveBlocks;

  	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, _mul_vecs, 0, 0);
  	realGridSize = (arr_len + blockSize - 1) / blockSize;

	hipDeviceProp_t prop;
  	hipGetDeviceProperties(&prop, 0);
	uint32_t smCount = prop.multiProcessorCount;
	hipError_t error = hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, _mul_vecs, blockSize, 0);
    if (error == hipSuccess)
    	realGridSize = maxActiveBlocks * smCount;

	_mul_vecs<<<realGridSize, blockSize>>>(a_arr, b_arr, c_arr, arr_len);
}

polynomial _polynomial_multiplication_on_fft(const polynomial& A, const polynomial& B)
{
    size_t n = get_power_of_two(A.deg + B.deg);
	polynomial C;
	C.deg = A.deg + B.deg;

	embedded_field* temp_memory1 = nullptr;
	embedded_field* temp_memory2 = nullptr;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void **)&temp_memory1, n * sizeof(embedded_field));
	cudaStatus = hipMalloc((void **)&temp_memory2, n * sizeof(embedded_field));
	cudaStatus = hipMalloc((void **)&C.coeffs, n * sizeof(embedded_field));
	
	hipMemcpy(temp_memory1, A.coeffs, A.deg * sizeof(embedded_field), hipMemcpyDeviceToDevice);
	hipMemset(temp_memory1 + A.deg, 0, (n - A.deg) *sizeof(embedded_field));
	hipMemcpy(temp_memory2, B.coeffs, B.deg * sizeof(embedded_field), hipMemcpyDeviceToDevice);
	hipMemset(temp_memory2 + B.deg, 0, (n - B.deg) *sizeof(embedded_field));

    FFT_DRIVER(temp_memory1, temp_memory1, n, false);
	FFT_DRIVER(temp_memory2, temp_memory2, n, false);
	
	_mul_vecs_driver(temp_memory1, temp_memory2, C.coeffs, n);	
	FFT_DRIVER(C.coeffs, C.coeffs, n, true);
	//_mul_elem_driver(C.coeffs, get_inv(n), n);

	hipFree(temp_memory1);
	hipFree(temp_memory2);

	return C;
}

#define POLY_MUL(X, Y) _polynomial_multiplication_on_fft(X, Y)


//these drivers are used only for test purposes
//-------------------------------------------------------------------------------------------------------------------------------------------------------------
//-------------------------------------------------------------------------------------------------------------------------------------------------------------
//-------------------------------------------------------------------------------------------------------------------------------------------------------------

void naive_FFT_test_driver(uint256_g* A, uint256_g* B, uint256_g* C, size_t arr_len)
{
	naive_fft_driver(reinterpret_cast<embedded_field*>(A), reinterpret_cast<embedded_field*>(C), arr_len);
}

void advanced_fft_test_driver(uint256_g* A, uint256_g* B, uint256_g* C, size_t arr_len)
{
	advanced_fft_driver(reinterpret_cast<embedded_field*>(A), reinterpret_cast<embedded_field*>(C), arr_len);
}
