#include "hip/hip_runtime.h"
#include "cuda_structs.h"

//FFT (we propose very naive realization)
//----------------------------------------------------------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------------------------------------------------------------------

//Sources of inspiration:
//http://www.staff.science.uu.nl/~bisse101/Articles/preprint1138.pdf
//https://cs.wmich.edu/gupta/teaching/cs5260/5260Sp15web/studentProjects/tiba&hussein/03278999.pdf
//http://users.umiacs.umd.edu/~ramani/cmsc828e_gpusci/DeSpain_FFT_Presentation.pdf
//http://www.bealto.com/gpu-fft_intro.html
//https://github.com/mmajko/FFT-cuda/blob/master/src/fft-cuda.cu
//Also have a loot at GPU gems

//NB: arr should be a power of two

//this is a field embedded into a group of points on elliptic curve

//because of FUNCKING NVIDIA CREW I'am unable to use inline asm here

struct embedded_field
{
	uint256_g rep_;

	DEVICE_FUNC explicit embedded_field(const uint256_g rep): rep_(rep) {}
	
	DEVICE_FUNC bool operator==(const embedded_field& other) const
	{
		return EQUAL(rep_, other.rep_);
	}

	DEVICE_FUNC bool operator!=(const embedded_field& other) const
	{
		return !EQUAL(rep_, other.rep_);
	}

	DEVICE_FUNC operator uint256_g() const
	{
		return rep_;
	}

	DEVICE_FUNC embedded_field operator-() const
	{
		if (!is_zero(rep_))
			return embedded_field(SUB(EMBEDDED_FIELD_P, rep_));
		else
			return *this;
	}

	//NB: for now we assume that highest possible limb bit is zero for the field modulus
	DEVICE_FUNC embedded_field& operator+=(const embedded_field& other)
	{
		rep_ = ADD(rep_, other.rep_);
		if (CMP(rep_, EMBEDDED_FIELD_P) >= 0)
			rep_ = SUB(rep_, EMBEDDED_FIELD_P);
		return *this;
	}

	DEVICE_FUNC embedded_field& operator-=(const embedded_field& other)
	{
		if (CMP(rep_, other.rep_) > 0)
			rep_ = SUB(rep_, other.rep_);
		else
		{
			uint256_g t = ADD(rep_, EMBEDDED_FIELD_P);
			rep_ = SUB(t, other.rep_);
		}
		return *this;
	}

	//here we mean montgomery multiplication

	DEVICE_FUNC embedded_field& operator*=(const embedded_field& other)
	{
		uint256_g T;
		uint256_g u = rep_;
		uint256_g v = other.rep_;

		#pragma unroll
		for (uint32_t j = 0; j < N; j++)
			T.n[j] = 0;

		uint32_t prefix_low = 0, prefix_high = 0, m;
		uint32_t high_word, low_word;

		#pragma unroll
		for (uint32_t i = 0; i < N; i++)
		{
			uint32_t carry = 0;
			#pragma unroll
			for (uint32_t j = 0; j < N; j++)
			{         
				low_word = device_long_mul(u.n[j], v.n[i], &high_word);
				low_word = device_fused_add(low_word, T.n[j], &high_word);
				low_word = device_fused_add(low_word, carry, &high_word);
				carry = high_word;
				T.n[j] = low_word;
			}

			//TODO: may be we actually require less space? (only one additional limb instead of two)
			prefix_high = 0;
			prefix_low = device_fused_add(prefix_low, carry, &prefix_high);

			m = T.n[0] * EMBEDDED_FIELD_N;
			low_word = device_long_mul(EMBEDDED_FIELD_P.n[0], m, &high_word);
			low_word = device_fused_add(low_word, T.n[0], &high_word);
			carry = high_word;

			#pragma unroll
			for (uint32_t j = 1; j < N; j++)
			{
				low_word = device_long_mul(EMBEDDED_FIELD_P.n[j], m, &high_word);
				low_word = device_fused_add(low_word, T.n[j], &high_word);
				low_word = device_fused_add(low_word, carry, &high_word);
				T.n[j-1] = low_word;
				carry = high_word;
			}

			T.n[N-1] = device_fused_add(prefix_low, carry, &prefix_high);
			prefix_low = prefix_high;
		}
		
		if (CMP(T, EMBEDDED_FIELD_P) >= 0)
		{
			//TODO: may be better change to inary version of sub?
			T = SUB(T, EMBEDDED_FIELD_P);
		}

		rep_ = T;
		return *this;
	}
		
	friend DEVICE_FUNC embedded_field operator+(const embedded_field& left, const embedded_field& right);
	friend DEVICE_FUNC embedded_field operator-(const embedded_field& left, const embedded_field& right);
	friend DEVICE_FUNC embedded_field operator*(const embedded_field& left, const embedded_field& right);
};
	

DEVICE_FUNC embedded_field operator+(const embedded_field& left, const embedded_field& right)
{
	embedded_field result(left);
	result += right;
	return result;
}

DEVICE_FUNC embedded_field operator-(const embedded_field& left, const embedded_field& right)
{
	embedded_field result(left);
	result -= right;
	return result;
}

DEVICE_FUNC embedded_field operator*(const embedded_field& left, const embedded_field& right)
{
	embedded_field result(left);
	result *= right;
	return result;
}

//commom FFT routines
//------------------------------------------------------------------------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------------------------------------------------
//------------------------------------------------------------------------------------------------------------------------------------------------------------

struct field_pair
{
	embedded_field a;
	embedded_field b;
};

DEVICE_FUNC field_pair __inline__ fft_buttefly(const embedded_field& x, const embedded_field& y, const embedded_field& root_of_unity)
{
	embedded_field temp = y * root_of_unity;
	return field_pair{ x + temp, x - temp};
}

DEVICE_FUNC embedded_field __inline__ get_root_of_unity(uint32_t index, uint32_t omega_idx_coeff = 1)
{
	embedded_field result(EMBEDDED_FIELD_R);
	uint32_t real_idx = index * omega_idx_coeff;
	for (unsigned k = 0; k < ROOTS_OF_UNTY_ARR_LEN; k++)
	{
		if (CHECK_BIT(real_idx, k))
			result *= embedded_field(EMBEDDED_FIELD_ROOTS_OF_UNITY[k]);
	}
	return result;	
}

struct geometry
{
    int gridSize;
    int blockSize;
};

template<typename T>
geometry find_suitable_geometry(T func, uint shared_memory_used, uint32_t smCount)
{
    int gridSize;
    int blockSize;
    int maxActiveBlocks;

    hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, func, shared_memory_used, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, func, blockSize, shared_memory_used);
    gridSize = maxActiveBlocks * smCount;

    return geometry{gridSize, blockSize};
}

//Naive FFT-realization
//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------

__global__ void __inline__ FFT_shuffle(embedded_field* __restrict__ input_arr, embedded_field* __restrict__ output_arr, uint32_t arr_len)
{
	uint32_t  tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < arr_len)
	{
		output_arr[tid] = input_arr[__brev(tid)];
		tid += blockDim.x * gridDim.x;
	}
}

__global__ void FFT_iteration(embedded_field* __restrict__ input_arr, embedded_field* __restrict__ output_arr, 
	uint32_t arr_len, uint32_t log_arr_len, uint32_t step)
{
	uint32_t  i = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t k = (1 << step);
	uint32_t l = 2 * k;
	while (i < arr_len / 2)
	{
		uint32_t first_index = l * (i / k) + (i % k);
		uint32_t second_index = first_index + k;

		uint32_t root_of_unity_index = (1 << (log_arr_len - step - 1)) * (i % l); 
		embedded_field omega = get_root_of_unity(root_of_unity_index);

		field_pair ops = fft_buttefly(input_arr[first_index], input_arr[second_index], omega);

		output_arr[first_index] = ops.a;
		output_arr[second_index] = ops.b;

		i += blockDim.x * gridDim.x;
	}
}

void fft_driver(embedded_field* __restrict__ input_arr, embedded_field* __restrict__ output_arr, uint32_t arr_len)
{
	//first check that arr_len is a power of 2

	uint log_arr_len = BITS_PER_LIMB - __builtin_clz(arr_len) - 1;
    assert(arr_len = (1 << log_arr_len));

	//find optimal geometry

	hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
	uint32_t smCount = prop.multiProcessorCount;

	geometry FFT_shuffle_geometry = find_suitable_geometry(FFT_shuffle, 0, smCount);
	geometry FFT_iter_geometry = find_suitable_geometry(FFT_iteration, 0, smCount);

	//allocate additional memory

	embedded_field* additional_device_memory = nullptr;
	hipError_t cudaStatus = hipMalloc((void **)&additional_device_memory, arr_len * sizeof(embedded_field));
	
	//FFT shuffle;

	embedded_field* temp_output_arr = (log_arr_len % 2 ? additional_device_memory : output_arr);
	embedded_field* temp_input_arr = (log_arr_len % 2 ? output_arr : additional_device_memory);
	FFT_shuffle<<<FFT_shuffle_geometry.gridSize, FFT_shuffle_geometry.blockSize>>>(input_arr, temp_output_arr, arr_len);
	
	//FFT main cycle

	for (uint32_t step = 0; step < log_arr_len; step++)
	{
		//swap input and iutput arrs

		embedded_field* swap_arr = temp_input_arr;
		temp_input_arr = temp_output_arr;
		temp_output_arr = swap_arr;
		
		FFT_iteration<<<FFT_iter_geometry.gridSize, FFT_iter_geometry.blockSize>>>(temp_input_arr, temp_output_arr, arr_len, log_arr_len, step);
	}

	//clean_up
	hipFree(additional_device_memory);
}


//Bellman FFT-realization
//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------

DEVICE_FUNC void _basic_serial_radix2_FFT(const embedded_field* input_arr, embedded_field* output_arr, size_t log_arr_len, size_t omega_idx_coeff)
{
	__shared__ embedded_field temp_arr[];

	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

	for(size_t i = tid; i < arr_len; i+= blockDim.x * gridDim.x;)
	{
		temp_arr[i] = input_arr[__brev(i)];
	}

	__syncthreads();
	
    for (size_t step = 1; step <= log_arr_len; ++step)
    {
        uint32_t i = tid;
		uint32_t k = (1 << step);
		uint32_t l = 2 * k;
		while (i < arr_len / 2)
		{
			uint32_t first_index = l * (i / k) + (i % k);
			uint32_t second_index = first_index + k;

			uint32_t omega_idx = (1 << (log_arr_len - step - 1)) * (i % l); 
			embedded_field omega = get_root_of_unity(omega_idx, omega_idx, coeff);

			field_pair ops = fft_buttefly(temp_arr[first_index], temp_arr[second_index], omega);

			temp_arr[first_index] = ops.a;
			temp_arr[second_index] = ops.b;

			i += blockDim.x * gridDim.x;
		}
		
		__syncthreads();
	}

	output_arr



		
    }
}

__kernel__ void _basic_parallel_radix2_FFT(const embedded_field* input_arr, embedded_field* output_arr, size_t log_arr_len, size_t log_num_subblocks)
{
    __shared__ embedded_field temp_arr[];

	assert( log_arr_len <= ROOTS_OF_UNTY_ARR_LEN, "the size of array is too large for FFT");
	size_t arr_len = 1 << log_arr_len;
	size_t omega_coeff = 1 << (ROOTS_OF_UNTY_ARR_LEN - log_arr_len);
	size_t L = 1 << (log_arr_len - log_num_subblocks);
	size_t NUM_SUBBLOCKS = 1 << log_num_subblocks;

	embdedded_field omega_step = get_root_of_unity(blockIdx.x * L, omega_coeff);
        
    for (size_t i = threadIdx.x; i < L; i+= blockDim.x)
    {
        embdedded_field omega_init = get_root_of_unity(blockIdx.x * threadIdx.x, omega_coeff);
		temp[i] = 0;
		for (size_t s = 0; s < NUM_SUBBLOCKS; ++s)
        {
            size_t idx = i + s * L
            temp_arr[i] += input_arr[idx] * omega_init;
            omega_init *= omega_step;
        }
	}

	__syncthreads();

	embedded_field omega = get_root_of_unity(NUM_SUBBLOCKS, omega_coeff);
	_basic_serial_radix2_FFT(temp_arr, temp_arr, log_arr_len, omega_idx_coeff);

	for (size_t i = threadIdx.x; i < L; i+= blockDim.x)
		output_arr[i * NUM_SUBBLOCKS + blockidx.x] = tmp_arr[i];
}

template<typename FieldT>
void _basic_parallel_radix2_FFT(std::vector<FieldT> &a, const FieldT &omega)
{
#ifdef MULTICORE
    const size_t num_cpus = omp_get_max_threads();
#else
    const size_t num_cpus = 1;
#endif
    const size_t log_cpus = ((num_cpus & (num_cpus - 1)) == 0 ? log2(num_cpus) : log2(num_cpus) - 1);

#ifdef DEBUG
    libff::print_indent(); printf("* Invoking parallel FFT on 2^%zu CPUs (omp_get_max_threads = %zu)\n", log_cpus, num_cpus);
#endif

    if (log_cpus == 0)
    {
        _basic_serial_radix2_FFT(a, omega);
    }
    else
    {
        _basic_parallel_radix2_FFT_inner(a, omega, log_cpus);
    }
}

//make the same things using shuffle instructions

//polynomial multiplication via FFT

template<typename FieldT>
void _polynomial_multiplication(std::vector<FieldT> &c, const std::vector<FieldT> &a, const std::vector<FieldT> &b)
{
    _polynomial_multiplication_on_fft(c, a, b);
}

template<typename FieldT>
void _polynomial_multiplication_on_fft(std::vector<FieldT> &c, const std::vector<FieldT> &a, const std::vector<FieldT> &b)
{
    const size_t n = libff::get_power_of_two(a.size() + b.size() - 1);
    FieldT omega = libff::get_root_of_unity<FieldT>(n);

    std::vector<FieldT> u(a);
    std::vector<FieldT> v(b);
    u.resize(n, FieldT::zero());
    v.resize(n, FieldT::zero());
    c.resize(n, FieldT::zero());

#ifdef MULTICORE
    _basic_parallel_radix2_FFT(u, omega);
    _basic_parallel_radix2_FFT(v, omega);
#else
    _basic_serial_radix2_FFT(u, omega);
    _basic_serial_radix2_FFT(v, omega);
#endif

    std::transform(u.begin(), u.end(), v.begin(), c.begin(), std::multiplies<FieldT>());

#ifdef MULTICORE
    _basic_parallel_radix2_FFT(c, omega.inverse());
#else
    _basic_serial_radix2_FFT(c, omega.inverse());
#endif

    const FieldT sconst = FieldT(n).inverse();
    std::transform(c.begin(), c.end(), c.begin(), std::bind1st(std::multiplies<FieldT>(), sconst));
    _condense(c);
}

