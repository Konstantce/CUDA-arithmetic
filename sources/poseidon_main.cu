#include "cuda_structs.h"
#include "poseidon.h"

#include <chrono>
#include <stdlib.h>

#include <stdint.h>
#include <vector>
#include <iostream>

#include <stdio.h>
#include <time.h>

# define BENCH_SIZE (1000000)


struct Geometry
{
    int grid_size;
    int block_size;
};


template<typename T>
Geometry find_optimal_geometry(T func)
{
    int gridSize;
    int blockSize;
    int maxActiveBlocks;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t smCount = prop.multiProcessorCount;

    hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, func, 0, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, func, blockSize, 0);
    gridSize = maxActiveBlocks * smCount;

    return Geometry{ gridSize, blockSize };
}


// inspired by "Hacker's delight"
uint32_t round_down_to_power_of_2(uint32_t x) 
{
    x = x | (x >> 1);
    x = x | (x >> 2);
    x = x | (x >> 4);
    x = x | (x >> 8);
    x = x | (x >> 16);
    
    return x - (x >> 1);
}


int main(int argc, char* argv[])
{
    embedded_field* host_arr = nullptr;
    embedded_field* device_arr = nullptr;
    hiprandState* dev_states = nullptr;

    int return_error_code = 0;

    std::chrono::high_resolution_clock::time_point start, end;
    std::int64_t duration;
    hipError_t cudaStatus;
    
    bool result = CUDA_init();
    if (!result)
    {
        fprintf(stderr, "error on cuda init");
        return_error_code = -1;
        goto Error;
    }
    get_device_info();

    std::cout << "RUNNING POSEIDON BENCHMARK with  " << BENCH_SIZE << " ELEMENTS" << std::endl << std::endl;

    cudaStatus = hipMalloc(&device_arr, BENCH_SIZE * sizeof(embedded_field));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc (device_memory) failed!\n");
        return_error_code = -1;
        goto Error;
    }

    // we generate all the random elements on the device 
    {
        Geometry rand_gm = find_optimal_geometry(gen_random_array_kernel<embedded_field>);
        
        cudaStatus = hipMalloc((void**)&dev_states, rand_gm.grid_size * rand_gm.block_size * sizeof(hiprandState));
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMalloc (devStates) failed!\n");
            return_error_code = -1;
            goto Error;
        }
        
        long ltime = time(NULL);
        unsigned int stime = (unsigned int)ltime / 2;
        srand(stime);

        gen_random_array_kernel << < rand_gm.grid_size, rand_gm.block_size >> > (device_arr, BENCH_SIZE, dev_states, rand());

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "random elements generator kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return_error_code = -1;
            goto Error;
        }
    }

    // although we run our randomizer generator on the device we want to simulate the "real" flow of execution:
    // cpu -> device -> cpu
    // that's why we copy created elements to CPU (and then we will copy them back, but benchmarking the time)

    host_arr = (embedded_field*)malloc(BENCH_SIZE * sizeof(embedded_field));
    
    cudaStatus = hipMemcpy(host_arr, device_arr, BENCH_SIZE * sizeof(embedded_field), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy (device -> host) failed!\n");
        return_error_code = -1;
        goto Error;
    }

    {
        start = std::chrono::high_resolution_clock::now();

        cudaStatus = hipMemcpy(device_arr, host_arr, BENCH_SIZE * sizeof(embedded_field), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy (host -> device) failed!\n");
            return_error_code = -1;
            goto Error;
        }

        end = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
        std::cout << "MEMCOPY: HOST -> DEVICE took " << std::dec << duration << "ns." << std::endl << std::endl;
    }

    {
        Geometry poseidon_gm = find_optimal_geometry(poseidon);     
        start = std::chrono::high_resolution_clock::now();

        std::cout << "POSEIDON GEOMETRY : (num_blocks, num_threads_per_block): (" << poseidon_gm.grid_size << ", " << poseidon_gm.block_size << ")" << std::endl << std::endl; 

        poseidon<< <poseidon_gm.grid_size, poseidon_gm.block_size >> > (device_arr, BENCH_SIZE);

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "poseidon kernel failed with error code %s\n", cudaStatus);
            return_error_code = -1;
            goto Error;
        }

        end = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
        std::cout << "POSEIDON on GPU took " << std::dec << duration << "ns." << std::endl << std::endl;
    }

    {
        start = std::chrono::high_resolution_clock::now();

        cudaStatus = hipMemcpy(host_arr, device_arr, BENCH_SIZE * sizeof(embedded_field), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy (device -> host) failed!\n");
            return_error_code = -1;
            goto Error;
        }

        end = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
        std::cout << "MEMCOPY: DEVICE -> HOST took " << std::dec << duration << "ns." << std::endl << std::endl;
    }

    {
        // here we construct merklee tree on GPU 

        // for benchmarking purposes only binary tree are supported
        assert(POSEIDON_TREE_COLLAPSING_FACTOR == 2);
        
        uint32_t tree_size = round_down_to_power_of_2(BENCH_SIZE); 
        std::cout << "POSEIDON MERKLE TREE CONSTRUCTION benchmark for size: " << tree_size << std::endl;
        
        Geometry poseidon_gm = find_optimal_geometry(poseidon_merkle_tree_construction_iteration);     
        start = std::chrono::high_resolution_clock::now();

        std::cout << "POSEIDON MERKLE TREE GEOMETRY : (num_blocks, num_threads_per_block): (" << poseidon_gm.grid_size << ", " << poseidon_gm.block_size << ")" << std::endl << std::endl; 

        while (tree_size > THREADS_PER_BLOCK * POSEIDON_TREE_COLLAPSING_FACTOR)
        {
            poseidon_merkle_tree_construction_iteration<< <poseidon_gm.grid_size, poseidon_gm.block_size >> >(device_arr, tree_size);
            tree_size /= POSEIDON_TREE_COLLAPSING_FACTOR;

            cudaStatus = hipDeviceSynchronize();
            if (cudaStatus != hipSuccess)
            {
                fprintf(stderr, "poseidon merkle tree construction failed with error code %s\n", cudaStatus);
                return_error_code = -1;
                goto Error;
            }
        }
        
        poseidon_merkle_tree_single_block << <1, THREADS_PER_BLOCK >> >(device_arr, tree_size);
        cudaStatus = hipDeviceSynchronize();
        
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "poseidon merkle tree construction failed (on last iteration) with error code %s\n", cudaStatus);
            return_error_code = -1;
            goto Error;
        }

        end = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
        std::cout << "POSEIDON MERKLE TREE CONSTRUCTION on GPU took " << std::dec << duration << "ns." << std::endl << std::endl;
    }

Error:
    hipFree(device_arr);
    hipFree(dev_states);

    free(host_arr);

    return return_error_code;
}

