#include "cuda_structs.h"
#include "poseidon.h"

#include <chrono>
#include <stdlib.h>

#include <stdint.h>
#include <vector>
#include <iostream>

#include <stdio.h>
#include <time.h>

# define BENCH_SIZE (1000000)


struct Geometry
{
    int grid_size;
    int block_size;
};


template<typename T>
Geometry find_optimal_geometry(T func)
{
    int gridSize;
    int blockSize;
    int maxActiveBlocks;

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t smCount = prop.multiProcessorCount;

    hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, func, 0, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, func, blockSize, 0);
    gridSize = maxActiveBlocks * smCount;

    return Geometry{ gridSize, blockSize };
}


int main(int argc, char* argv[])
{
    embedded_field* host_arr = nullptr;
    embedded_field* device_arr = nullptr;
    hiprandState* dev_states = nullptr;

    int return_error_code = 0;

    std::chrono::high_resolution_clock::time_point start, end;
    std::int64_t duration;
    hipError_t cudaStatus;
    
    bool result = CUDA_init();
    if (!result)
    {
        fprintf(stderr, "error on cuda init");
        return_error_code = -1;
        goto Error;
    }
    get_device_info();

    std::cout << "RUNNING POSEIDON BENCHMARK with  " << BENCH_SIZE << " ELEMENTS" << std::endl << std::endl;

    cudaStatus = hipMalloc(&device_arr, BENCH_SIZE * sizeof(embedded_field));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc (device_memory) failed!\n");
        return_error_code = -1;
        goto Error;
    }

    // we generate all the random elements on the device 
    {
        Geometry rand_gm = find_optimal_geometry(gen_random_array_kernel<embedded_field>);
        
        cudaStatus = hipMalloc((void**)&dev_states, rand_gm.grid_size * rand_gm.block_size * sizeof(hiprandState));
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMalloc (devStates) failed!\n");
            return_error_code = -1;
            goto Error;
        }
        
        long ltime = time(NULL);
        unsigned int stime = (unsigned int)ltime / 2;
        srand(stime);

        gen_random_array_kernel << < rand_gm.grid_size, rand_gm.block_size >> > (device_arr, BENCH_SIZE, dev_states, rand());

        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "random elements generator kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return_error_code = -1;
            goto Error;
        }
    }

    // although we run our randomizer generator on the device we want to simulate the "real" flow of execution:
    // cpu -> device -> cpu
    // that's why we copy created elements to CPU (and then we will copy them back, but benchmarking the time)

    host_arr = (embedded_field*)malloc(BENCH_SIZE * sizeof(embedded_field));
    
    cudaStatus = hipMemcpy(host_arr, device_arr, BENCH_SIZE * sizeof(embedded_field), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy (device -> host) failed!\n");
        return_error_code = -1;
        goto Error;
    }

    {
        start = std::chrono::high_resolution_clock::now();

        cudaStatus = hipMemcpy(device_arr, host_arr, BENCH_SIZE * sizeof(embedded_field), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy (host -> device) failed!\n");
            return_error_code = -1;
            goto Error;
        }

        end = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
        std::cout << "MEMCOPY: HOST -> DEVICE took " << std::dec << duration << "ns." << std::endl << std::endl;
    }

    {
        Geometry poseidon_gm = find_optimal_geometry(poseidon);     
        start = std::chrono::high_resolution_clock::now();

        poseidon<< <poseidon_gm.grid_size, poseidon_gm.block_size >> > (device_arr, BENCH_SIZE);

        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "poseidon kerkem failed with error code!\n", cudaStatus);
            return_error_code = -1;
            goto Error;
        }

        end = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
        std::cout << "POSEIDON on GPU took " << std::dec << duration << "ns." << std::endl << std::endl;
    }

    {
        start = std::chrono::high_resolution_clock::now();

        cudaStatus = hipMemcpy(host_arr, device_arr, BENCH_SIZE * sizeof(embedded_field), hipMemcpyDeviceToHost);
        if (cudaStatus != hipSuccess)
        {
            fprintf(stderr, "hipMemcpy (device -> host) failed!\n");
            return_error_code = -1;
            goto Error;
        }

        end = std::chrono::high_resolution_clock::now();
        duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
        std::cout << "MEMCOPY: DEVICE -> HOST took " << std::dec << duration << "ns." << std::endl << std::endl;
    }

Error:
    hipFree(device_arr);
    hipFree(dev_states);

    free(host_arr);

    return return_error_code;
}

