#include "hip/hip_runtime.h"
#include "cuda_structs.h"

//how to reduce:
//DEVICE_FUNC uint256_g mont_mul_256_asm_CIOS(const uint256_g& u, const uint256_g& v)

//Sime useful links:

//What about dealing with tail effects?
//https://devblogs.nvidia.com/cuda-pro-tip-minimize-the-tail-effect/

//efficient finite field library:
//http://mpfq.gforge.inria.fr/doc/index.html

//additional tricks for synchronization:
//https://habr.com/ru/post/151897/

//Why we do coalesce memory accesses:
//https://devblogs.nvidia.com/how-access-global-memory-efficiently-cuda-c-kernels/

//check strided kernels: look at XMP as a source of inspiration
//TODO: investigate why it works

//-----------------------------------------------------------------------------------------------------------------------------------------------------
//-----------------------------------------------------------------------------------------------------------------------------------------------------
//-----------------------------------------------------------------------------------------------------------------------------------------------------

//strided version of multiplication

#define GEOMETRY 128
#define ROUND_UP(n,d) (((n)+(d)-1)/(d)*(d))
#define DIV_ROUND_UP(n,d) (((n)+(d)-1)/(d))

struct geometry2
{
    int gridSize;
    int blockSize;
};

template<typename T>
geometry2 find_geometry2(T func, uint shared_memory_used, uint32_t smCount)
{
    int gridSize;
    int blockSize;
    int maxActiveBlocks;

    hipOccupancyMaxPotentialBlockSize(&gridSize, &blockSize, func, shared_memory_used, 0);
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, func, blockSize, shared_memory_used);
    gridSize = maxActiveBlocks * smCount;

    return geometry2{gridSize, blockSize};
}

__global__ void xmpC2S_kernel(uint32_t count, uint32_t limbs, uint32_t stride, const uint32_t * in, uint32_t * out)
{
    //outer dimension = count
    //inner dimension = limbs

    //read strided in inner dimension`
    //write coalesced in outer dimension
    for(uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x*gridDim.x)
    {
        for(uint32_t j=blockIdx.y * blockDim.y + threadIdx.y; j < limbs; j += blockDim.y * gridDim.y)
        {
            out[j*stride + i] = in[i*limbs + j];
        }
    }
}

inline void xmpC2S(uint32_t count, const uint32_t * in, uint32_t * out)
{
    dim3 threads, blocks;
    uint32_t limbs = N;
    //round up to 128 bше boundarн
    uint32_t stride = ROUND_UP(count, 32);  

    //target 128 threads
    threads.x = MIN(32, count);
    threads.y = MIN(DIV_ROUND_UP(128, threads.x), limbs);

    blocks.x = DIV_ROUND_UP(count, threads.x);
    blocks.y=DIV_ROUND_UP(limbs, threads.y);

    //convert from climbs to slimbs
    xmpC2S_kernel<<<blocks,threads>>>(count, limbs, stride, in, out);
}

__global__ void xmpS2C_kernel(uint32_t count, uint32_t limbs, uint32_t stride, const uint32_t * in, uint32_t * out)
{
    //outer dimension = limbs
    //inner dimension = N

    //read strided in inner dimension
    //write coalesced in outer dimension
    for(uint32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < limbs; i += blockDim.x * gridDim.x)
    {
        for(uint32_t j = blockIdx.y * blockDim.y + threadIdx.y; j < count; j + =blockDim.y * gridDim.y)
        {
            out[j * limbs + i] = in[ i * stride + j];
        }
    }
}

inline void xmpS2C(uint32_t count, const uint32_t * in, uint32_t * out)
{
    dim3 threads, blocks;
    uint32_t limbs = N;
    //round up to 128 bше boundarн

    //target 128 threads
    threads.x = MIN(32, limbs);
    threads.y = MIN(DIV_ROUND_UP(128, threads.x), count);

    blocks.x=DIV_ROUND_UP(count, threads.x);
    blocks.y=DIV_ROUND_UP(limbs, threads.y);

    //convert from climbs to slimbs
    xmpS2C_kernel<<<blocks,threads>>>(count, limbs, stride, in, out);
}

#define STRIDED_MONT_MUL_TEST(func_name) \
__global__ void func_name##_kernel_strided(uint32_t* a_arr, uint32_t* b_arr, uint32_t* c_arr, size_t count)\
{\
    uint32_t stride = ROUND_UP(count, 32);\
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;\
    uint256_g A, B, C;\
    while (tid < count)\
    {\
        uint32_t* a_data = a_arr + tid;\
        uint32_t* b_data = b_arr + tid;\
        uint32_t* c_data = c_arr + tid;\
        #pragma unroll\
        for(uint32_t index = 0; index < N; index++)\
        {\
            A.n[index] = a_data[index * stride];\
            B.n[index] = b_data[index * stride];\
        }\
\
        C = func_name(A, B);\
\
        #pragma unroll\
        for(uint32_t index = 0; index < N; index++)\
        {\
            c_data[index * stride]= C[index];
        }\
    }\
}\
\
void func_name##_driver_strided(uint256_g* a_arr, uint256_g* b_arr, uin256_g* c_arr, size_t count)\
{\       
    hipDeviceProp_t prop;\
    hipGetDeviceProperties(&prop, 0);\
    uint32_t smCount = prop.multiProcessorCount;\   
    geometry2 geometry = find_geometry2(T func, 0, uint32_t smCount);\
\
    std::cout << "Grid size: " << geometry.gridSize << ",  blockSize: " << geometry.blockSize << std::endl;\
    func_name##_kernel<<<geometry.gridSize, geometry.blockSize>>>(reinterpet_cast<uint32_t*>(a_arr), reinterpet_cast<uint32_t*>(b_arr),\ 
        reinterpet_cast<uint32_t*>(c_arr), count);\
}

//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------

//warp-based long multiplication and montgomery multiplication

#define THREADS_PER_MUL 8

#define UNROLLED_CYCLE_ITER(idx) \
"shfl.sync.idx.b32  x, b, idx, 8;" \
"mad.lo.cc.u32 v, a, x, v;\n\t" \
"madc.hi.cc.u32 u, a, x, u;\n\t" \
"addc.u32 t 0, 0;\n\t" \
"shfl.sync.up.b32  v, v, 1, 8;" \
"shfl.sync.up.b32  c, c, 1, 8;" \
"and.type y, %laneid, 7;\n\t" \
"setp.eq.u32  p, y, 7;\n\t"
"@p {\n\t" \
"mov.u32 c v;\n\t" \
"mov.u32 v 0;\n\t"
"}\n\t" \
"add.cc.u32 v, u, v;\n\t" \
"addc.u32 u, t, 0;\n\t"

//The following implementation is based on paper "A Warp-synchronous Implementation for Multiple-length Multiplication on the GPU"

DEVICE_FUNC uint64_t asm_mul_warp_based(uint32_t A, const uint32_t B)
{
    uint64_t res;
    
    asm(    "{\n\t"  
            ".reg .u32 a, b, x, y, u, v, c, t;\n\t"
            ".reg .pred p;\n\t"

            // "ld.global.u32 a, [A + %laneid];\n\t"
            // "ld.global.u32 b, [B + %laneid];\n\t"
            "mov.b32 a, %0;\n\t"
            "mov.b32 b, %1;\n\t"

            "mov.u32 u, 0;\n\t"
            "mov.u32 v, 0;\n\t"
            "mov.u32 c, 0;\n\t"

            UNROLLED_CYCLE_ITER(0)
            UNROLLED_CYCLE_ITER(1)
            UNROLLED_CYCLE_ITER(2)
            UNROLLED_CYCLE_ITER(3)
            UNROLLED_CYCLE_ITER(4)
            UNROLLED_CYCLE_ITER(5)
            UNROLLED_CYCLE_ITER(6)
            UNROLLED_CYCLE_ITER(7)

            "L1:\n\t"
            "setp.eq.u32   p, ne, 0;\n\t"
            "vote.sync.any.pred  p, p, 8;\n\t"
            "@!p bra L2;\n\t"
            "shfl.sync.down.b32  u, u, 1, 8;\n\t"
            "add.cc.u32 v, v, u;\n\t"
            "addc.u32 u, 0, 0;\n\t"
            "bra L1;\n\t"
            
            "L2:\n\t"

            // "st.global.u32 [OUT + %laneid], c;\n\t"
            // "st.global.u32 [OUT + %laneid + 8], v;\n\t"
            "mov.b64 %2, {c, v};}\n\t"  
            : "=l"(res) : "r"(A), "r"(B));
    
    return res;
}

//I bet there must be a faster solution!
//TODO: have a look at https://ieeexplore.ieee.org/document/5669278

__global__ void warp_based_mul_kernel(const uint256_g* a_arr, const uint256_g* b_arr, uint256_g* c_arr, size_t arr_len)
{
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    size_t idx = tid / THREADS_PER_MUL;
    size_t lane = tid % THREADS_PER_MUL;
	while (idx < arr_len)
	{
		uint32_t A = a_arr[idx].n[lane];
        uint32_t B = b_arr[idx].n[lane];
        uint64_t C = asm_mul_warp_based(A, B, C);

        c_arr[idx].n[lane] = C.low;
        c_arr[idx].n[lane + N] = C.high;

		tid += (blockDim.x * gridDim.x;) / THREADS_PER_MUL;
	}
}

void warp_based_mul_driver(const uint256_g* a_arr, const uint256_g* b_arr, uint256_g* c_arr, size_t arr_len)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t smCount = prop.multiProcessorCount;   
    geometry2 geometry = find_geometry2(warp_based_mul_kernel, 0, uint32_t smCount);

    std::cout << "Grid size: " << geometry.gridSize << ",  blockSize: " << geometry.blockSize << std::endl;
    warp_based_mul_kernel<<<geometry.gridSize, geometry.blockSize>>>(a_arr, b_arr, c_arr, arr_len);
}

//LARGE REDC Montgomety mul

DEVICE_FUNC __inline__ void long_in_place_add(uint32_t& A, uint32_t B, uint32_t& carry)
{
     asm(   "{\n\t"  
            "add.cc.u32 %0, %0, %2;\n\t"
            "addc.u32 %1, 0, 0;\n\t" 
            : "+r"(A), "=r"(carry) : "r"(B));
}

DEVICE_FUNC __inline__ void long_in_place_sub(uint32_t& A, uint32_t B, uint32_t& carry)
{
     asm(   "{\n\t"  
            "sub.cc.u32 %0, %0, %2;\n\t"
            "addc.u32 %1, 0, 0;\n\t" 
            : "+r"(A), "=r"(carry) : "r"(B));
}

#define MAX_UINT32_VAL 0xffffffff

DEVICE_FUNC __inline__ bool CHECK_CONDITIONS(uint32_t cond1_mask, uint32_t cond2_mask, uint32_t warp_idx)
{
    return ( __clz((cond2_mask << ((3 - warp_idx) * THREADS_PER_MUL)) ^ MAX_UINT32_VAL) >= __clz(cond1_mask << ((3 - warp_idx) * THREADS_PER_MUL)));
}

DEVICE_FUNC void mont_mul_warp_based(const uint256_g& A, const uint256_g& B, uint256_g& OUT)
{
    // T = A * B
    //m = ((T mod R) * N) mod R
    //t = (T + m * p) / R
    //if t >= N then t = t - N
       
    size_t lane = tid % THREADS_PER_MUL;
    size_t warp_idx = tid / THREADS_PER_MUL;
    uint32_t mask = (THREADS_PER_MUL - 1) << (warp_idx * THREADS_PER_MUL);

    uint64_t temp1 = asm_mul_warp_based(A.n[lane], B.n[lane]);
    uint64_t temp2 = asm_mul_warp_based(temp1.low, BASE_FIELD_N_LARGE.n[lane]);
    temp2 = asm_mul_warp_based(temp2, BASE_FIELD_P.n[lane]);

    //adding higher 8-words part
    uint32_t carry = 0;
    long_in_place_add(temp1.high, temp2.high, carry);

    //we are going to check if there is overflow from lower 8-words part
    uint32_t sum = temp1.low + temp2.low;
    uint32_t cond1_mask = __ballot_sync(mask, sum < temp1.low);
    uint32_t cond2_mask = _ballot_sync(mask, sum == MAX_UINT32_VAL);
    if (lane == THREADS_PER_MUL - 1)
        carry = (uint32_t)CHECK_CONDITIONS(cond1, cond2, warp_idx);

    //propagate carry
    while (__any_sync(mask, carry != 0))
    {
        carry = shfl_up_sync(mask, carry, 1, THREADS_PER_MUL);
        long_in_place_add(temp1.high, carry, carry);
        if (lane == THREADS_PER_MUL - 1)
            carry = 0;
    }

    //now temp1.high holds t, compare t with N:
    cond1 = __ballot_sync(mask, temp1.high > BASE_FIELD_P.n[lane]);
    cond2 = __ballot_sync(mask, temp1.high == BASE_FIELD_P.n[lane]);
    if (CHECK_CONDITIONS(cons1, cond2, warp_idx))
    {
        //lane based substraction
        long_in_place_sub(temp1.high, BASE_FIELD_P.n[lane], carry);

        //propagate borrow
        while (__any_sync(mask, carry != 0))
        {
            carry = shfl_up_sync(mask, carry, 1, THREADS_PER_MUL);
            long_in_place_sub(temp1.high, carry, carry);
  
        }
    }

    OUT.n[lane] = temp1.high;
}

__global__ void warp_based_mont_mul_kernel(const uint256_g* a_arr, const uint256_g* b_arr, uint256_g* c_arr, size_t arr_len)
{
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    size_t idx = tid / THREADS_PER_MUL;
	while (idx < arr_len)
	{
		uint256_g& A = a_arr[tid];
        uint256_g& B = b_arr[tid];
        uint256_g& C = c_arr[tid];

        mont_mul_warp_based(A, B, C);
		tid += (blockDim.x * gridDim.x;) / THREADS_PER_MUL;
	}
}

void warp_based_mont_mul_driver(const uint256_g* a_arr, const uint256_g* b_arr, uint256_g* c_arr, size_t arr_len)
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    uint32_t smCount = prop.multiProcessorCount;   
    geometry2 geometry = find_geometry2(warp_based_mont_mul_kernel, 0, uint32_t smCount);

    std::cout << "Grid size: " << geometry.gridSize << ",  blockSize: " << geometry.blockSize << std::endl;
    warp_based_mont_mul_kernel<<<geometry.gridSize, geometry.blockSize>>>(a_arr, b_arr, c_arr, arr_len);
}

//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------

//warp-based montgomery multiplication and elliptic curve point addition

//16 threads are used to calculate one operation

#define THREADS_PER_ECC_ADD 16

DEVICE_FUNC __inline__ bool is_leader_lane()
{
    return (threadIdx.x % THREADS_PER_ECC_ADD == 0);
}

DEVICE_FUNC __inline__ uint256_g& subwarp_chooser()



DEVICE_FUNC void ECC_add_proj_warp_based(const ec_point* A, const ec_point* B, ec_point* C)
{
    uint32_t exit_flag = 0;

    if (is_leader_lane())
    {
        if (is_infinity(*A))
        {
            *C = *B;
            exit_flag = 1;
        }
	    else if (is_infinity(*B))
        {
		    *C = *A;
            exit_flag = 1;
        }
    }

    exit_flag = __shfl_sync(0xFFFFFFFF, exit_flag, 0, THREADS_PER_ECC_ADD);
    if (exit_flag)
        return;

    uint32_t x, y;

	uint256_g U1, U2, V1, V2;
	U1 = MONT_MUL(left.z, right.y);
	U2 = MONT_MUL(left.y, right.z);
	V1 = MONT_MUL(left.z, right.x);
	V2 = MONT_MUL(left.x, right.z);

	ec_point res;

	if (EQUAL(V1, V2))
	{
		if (!EQUAL(U1, U2))
			return point_at_infty();
		else
			return  ECC_DOUBLE_PROJ(left);
	}

	uint256_g U = FIELD_SUB(U1, U2);
	uint256_g V = FIELD_SUB(V1, V2);
	uint256_g W = MONT_MUL(left.z, right.z);
	uint256_g Vsq = MONT_SQUARE(V);
	uint256_g Vcube = MONT_MUL(Vsq, V);

	uint256_g temp1, temp2;
	temp1 = MONT_SQUARE(U);
	temp1 = MONT_MUL(temp1, W);
	temp1 = FIELD_SUB(temp1, Vcube);
	temp2 = MONT_MUL(BASE_FIELD_R2, Vsq);
	temp2 = MONT_MUL(temp2, V2);
	uint256_g A = FIELD_SUB(temp1, temp2);
	res.x = MONT_MUL(V, A);

	temp1 = MONT_MUL(Vsq, V2);
	temp1 = FIELD_SUB(temp1, A);
	temp1 = MONT_MUL(U, temp1);
	temp2 = MONT_MUL(Vcube, U2);
	res.y = FIELD_SUB(temp1, temp2);

	res.z = MONT_MUL(Vcube, W);
	return res;

}

__global__ void ECC_add_proj_warp_based_kernel(const ec_point* a_arr, const ec_point* b_arr, ec_point *c_arr, size_t arr_len)
{
	size_t tid = (threadIdx.x + blockIdx.x * blockDim.x) / THREADS_PER_OP;
	while (tid < arr_len)
	{
		ECC_add_proj_warp_based(a_arr + tid, b_arr + tid, c_arr + tid);
		tid += (blockDim.x * gridDim.x) / 16;
	}
}
    
DEVICE_FUNC ec_point ECC_ADD_PROJ(const ec_point& left, const ec_point& right)
{
	if (is_infinity(left))
		return right;
	if (is_infinity(right))
		return left;

	uint256_g U1, U2, V1, V2;
	U1 = MONT_MUL(left.z, right.y);
	U2 = MONT_MUL(left.y, right.z);
	V1 = MONT_MUL(left.z, right.x);
	V2 = MONT_MUL(left.x, right.z);

	ec_point res;

	if (EQUAL(V1, V2))
	{
		if (!EQUAL(U1, U2))
			return point_at_infty();
		else
			return  ECC_DOUBLE_PROJ(left);
	}

	uint256_g U = FIELD_SUB(U1, U2);
	uint256_g V = FIELD_SUB(V1, V2);
	uint256_g W = MONT_MUL(left.z, right.z);
	uint256_g Vsq = MONT_SQUARE(V);
	uint256_g Vcube = MONT_MUL(Vsq, V);

	uint256_g temp1, temp2;
	temp1 = MONT_SQUARE(U);
	temp1 = MONT_MUL(temp1, W);
	temp1 = FIELD_SUB(temp1, Vcube);
	temp2 = MONT_MUL(BASE_FIELD_R2, Vsq);
	temp2 = MONT_MUL(temp2, V2);
	uint256_g A = FIELD_SUB(temp1, temp2);
	res.x = MONT_MUL(V, A);

	temp1 = MONT_MUL(Vsq, V2);
	temp1 = FIELD_SUB(temp1, A);
	temp1 = MONT_MUL(U, temp1);
	temp2 = MONT_MUL(Vcube, U2);
	res.y = FIELD_SUB(temp1, temp2);

	res.z = MONT_MUL(Vcube, W);
	return res;
}

//Transmit phase: 2 -> 1, 4 -> 3

//1
uint256_g U = FIELD_SUB(U1, U2);
//2
uint256_g V = FIELD_SUB(V1, V2);

uint256_g temp1, temp2;

//1
uint256_g Vsq = MONT_SQUARE(V);
//2
temp1 = MONT_SQUARE(U);
//3
uint256_g W = MONT_MUL(left.z, right.z);

//1
uint256_g Vcube = MONT_MUL(Vsq, V);
//2
temp1 = MONT_MUL(temp1, W);
//3
temp2 = MONT_MUL(BASE_FIELD_R2, Vsq);
//4
tempx = MONT_MUL(Vsq, V2);

//1
temp2 = MONT_MUL(temp2, V2);
//2
temp1 = FIELD_SUB(temp1, Vcube);
//3
tempg = MONT_MUL(Vcube, U2);
//4
res.z = MONT_MUL(Vcube, W);

uint256_g A = FIELD_SUB(temp1, temp2);
res.x = MONT_MUL(V, A);


temp1 = FIELD_SUB(tempx, A);
temp1 = MONT_MUL(U, temp1);

res.y = FIELD_SUB(temp1, tempg);


return res;
}

DEVICE_FUNC ec_point ECC_DOUBLE_PROJ(const ec_point& pt)
{
	if (is_zero(pt.y) || is_infinity(pt))
		return point_at_infty();
	else
	{
		uint256_g temp, temp2;
		uint256_g W, S, B, H, S2;
		ec_point res;

#ifdef BN256_SPECIFIC_OPTIMIZATION
 		temp = MONT_SQUARE(pt.x);
 		W = MONT_MUL(temp, R3_g);
#else
 		temp = MONT_SQUARE(pt.x);
 		temp = MONT_MUL(temp, BASE_FIELD_R3);
 		temp2 = MONT_SQUARE(pt.z);
 		temp2 = MONT_MUL(temp2, CURVE_A_COEFF);
 		W = FIELD_ADD(temp, temp2);
#endif
 		S = MONT_MUL(pt.y, pt.z);
		temp = MONT_MUL(pt.x, pt.y);
 		B = MONT_MUL(temp, S);
		res.x = W;

 		temp = MONT_SQUARE(W);
 		temp2 = MONT_MUL(BASE_FIELD_R8, B);
 		H = FIELD_SUB(temp, temp2);

 		temp = MONT_MUL(BASE_FIELD_R2, H);
 		res.x = MONT_MUL(temp, S);
		
 		//NB: here result is also equal to one of the operands and hence may be reused!!!
 		//NB: this is in fact another possibility for optimization!
 		S2 = MONT_SQUARE(S);
 		temp = MONT_MUL(BASE_FIELD_R4, B);
 		temp = FIELD_SUB(temp, H);
 		temp = MONT_MUL(W, temp);
		
 		temp2 = MONT_SQUARE(pt.y);
 		temp2 = MONT_MUL(BASE_FIELD_R8, temp2);
 		temp2 = MONT_MUL(temp2, S2);
 		res.y = FIELD_SUB(temp, temp2);

 		temp = MONT_MUL(BASE_FIELD_R8, S);
 		res.z = MONT_MUL(temp, S2);

		return res;
	}
}

//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------
//--------------------------------------------------------------------------------------------------------------------------------------------------------

//test exponentiation: one operation per warp

//we do also implement Montgomery ladder algorithm

#define MONTGOMERY_LADDER(SUFFIX) \
DEVICE_FUNC ec_point ecc_mont_ladder_exp##SUFFIX(const ec_point& pt, const uint256_g& power)\
{\
    ec_point R0 = point_at_infty();\
    ec_point R1 = pt;\
    for (int i = N_BITLEN - 1; i >= 0; i--)\
    {\
        bool flag = get_bit(power, i);\
        ec_point& Q = (flag ? R0 : R1);\
        ec_point& T = (flag ? R1 : R0);\
\
        Q = ECC_ADD##SUFFIX(Q, R);\
        T = ECC_DOUBLE##SUFFIX(T);\
    }\
\
    return R0;\
}
    
MONTGOMERY_LADDER(_PROJ)
MONTGOMERY_LADDER(_JAC)

#define EXP_ONE_OP_PER_WRAP(func_name) \
__global__ void func_name##_kernel_per_warp(const ec_point* a_arr, const uint256_g* b_arr, ec_point* c_arr, size_t count)\
{\
    size_t tid = (threadIdx.x + blockIdx.x * blockDim.x;) / WARP_SIZE\
	while (tid < arr_len)\
	{\
		if (threadIdx.x % WARP_SIZE == 0)\
            c_arr[tid] = func_name(a_arr[tid], b_arr[tid]);\
		tid += (blockDim.x * gridDim.x;) / WARP_SIZE\
	}\
}\
\
void func_name##_driver_per_warp(const ec_point* a_arr, const uint256_g* b_arr, ec_point* c_arr, size_t count)\
{\       
    hipDeviceProp_t prop;\
    hipGetDeviceProperties(&prop, 0);\
    uint32_t smCount = prop.multiProcessorCount;\   
    geometry2 geometry = find_geometry2(func_name##_kernel_per_warp, 0, uint32_t smCount);\
\
    std::cout << "Grid size: " << geometry.gridSize << ",  blockSize: " << geometry.blockSize << std::endl;\
    func_name##_kernel_per_warp<<<geometry.gridSize, geometry.blockSize>>>(a_arr, b_arr, c_arr, count);\
}


