#include "hip/hip_runtime.h"
#include "cuda_structs.h"

#include <iostream>

//Various algorithms for simultaneous multiexponentiation: naive approach and Pippenger algorithm
//naive approach was widely inspired by https://devblogs.nvidia.com/faster-parallel-reductions-kepler/
//---------------------------------------------------------------------------------------------------------------------------------------------------------
//---------------------------------------------------------------------------------------------------------------------------------------------------------
//---------------------------------------------------------------------------------------------------------------------------------------------------------

//There are four versions using naive approach:
//1) using warp level reduction and atomics
//2) using block level reduction and atomics
//3) using block level reduction and recursion

//TODO: it seems that the best way is to combine these approaches, e.g. do several levels of atomic add, than block reduce - there is a vast field
//for experiements

//TODO: implement using warp level reduction and recursion

//TODO: implement version with cooperative groups

//TODO: implement approach using CUB library: http://nvlabs.github.io/cub/index.html

//we have implemented vectorized loads inspired by: https://devblogs.nvidia.com/cuda-pro-tip-increase-performance-with-vectorized-memory-access/

//Useful miscellaneous functions
//-----------------------------------------------------------------------------------------------------------------------------------------------------------

DEVICE_FUNC inline void __shfl_down(const ec_point& in_var, ec_point& out_var, unsigned int offset, int width=32)
{
    //ec_point = 3 * 8  = 24 int = 6 int4
    const int4* a = reinterpret_cast<const int4*>(&in_var);
    int4* b = reinterpret_cast<int4*>(&out_var);

    for (unsigned i = 0; i < 6; i++)
    {
        b[i].x = __shfl_down_sync(a[i].x, offset, width);
        b[i].y = __shfl_down_sync(a[i].y, offset, width);
        b[i].z = __shfl_down_sync(a[i].z, offset, width);
        b[i].w = __shfl_down_sync(a[i].w, offset, width);
    }
}

DEVICE_FUNC inline ec_point warpReduceSum(ec_point val)
{
    for (int offset = WARP_SIZE/2; offset > 0; offset /= 2)
    { 
        ec_point temp;
        __shfl_down(val, temp, offset);
        val = ECC_ADD(val, temp);
    }
           
    return val;
}

DEVICE_FUNC inline ec_point blockReduceSum(ec_point val)
{
    // Shared mem for 32 partial sums
    static __shared__ ec_point shared[WARP_SIZE]; 
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    // Each warp performs partial reduction
    val = warpReduceSum(val);     

    // Write reduced value to shared memory
    if (lane==0)
        shared[wid]=val; 

    // Wait for all partial reductions
    __syncthreads();              

    //read from shared memory only if that warp existed
    val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : point_at_infty();

    //Final reduce within first warp
    if (wid == 0)
        val = warpReduceSum(val); 

    return val;
}


//1) using warp level reduction and atomics
//---------------------------------------------------------------------------------------------------------------------------------------------------

__global__ void naive_multiexp_kernel_warp_level_atomics(affine_point* point_arr, uint256_g* power_arr, ec_point* out, size_t arr_len, int* mutex)
{
	ec_point acc = point_at_infty();
    
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < arr_len)
	{   
        ec_point x = ECC_EXP(point_arr[tid], power_arr[tid]);
        acc = ECC_ADD(acc, x);
        tid += blockDim.x * gridDim.x;
	}

    acc = warpReduceSum(acc);
 
    if ((threadIdx.x & (warpSize - 1)) == 0)
    {
        while (atomicCAS(mutex, 0, 1) != 0);
          
        *out = ECC_ADD(*out, acc);
       
        atomicExch(mutex, 0);
    }  
}

void naive_multiexp_kernel_warp_level_atomics_driver(affine_point* point_arr, uint256_g* power_arr, ec_point* out, size_t arr_len)
{
	int blockSize;
  	int minGridSize;
  	int realGridSize;

    int* mutex;
    hipMalloc((void**)&mutex, sizeof(int));
    hipMemset(mutex, 0, sizeof(int));

  	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, naive_multiexp_kernel_warp_level_atomics, 0, 0);
    realGridSize = (arr_len + blockSize - 1) / blockSize;

	std::cout << "Grid size: " << realGridSize << ",  min grid size: " << minGridSize << ",  blockSize: " << blockSize << std::endl;

    //create point at infty and copy it to output arr

    ec_point point_at_infty = { 
        {0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000},
        {0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000001},
        {0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000}
    };

    hipMemcpy(out, &point_at_infty, sizeof(ec_point), hipMemcpyHostToDevice);

	naive_multiexp_kernel_warp_level_atomics<<<realGridSize, blockSize>>>(point_arr, power_arr, out, arr_len, mutex);

    hipFree(mutex);
}

//2) using block level reduction and atomics
//---------------------------------------------------------------------------------------------------------------------------------------------------

__global__ void naive_multiexp_kernel_block_level_atomics(affine_point* point_arr, uint256_g* power_arr, ec_point* out, size_t arr_len, int* mutex)
{
    ec_point acc = point_at_infty();
    
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < arr_len)
	{   
        ec_point x = ECC_EXP(point_arr[tid], power_arr[tid]);
        acc = ECC_ADD(acc, x);
        tid += blockDim.x * gridDim.x;
	}

    acc = blockReduceSum(acc);
    if (threadIdx.x == 0)
    {
        while (atomicCAS(mutex, 0, 1) != 0);
        *out = ECC_ADD(*out, acc);
        atomicExch(mutex, 0);  
    }
}

void naive_multiexp_kernel_block_level_atomics_driver(affine_point* point_arr, uint256_g* power_arr, ec_point* out, size_t arr_len)
{
	int blockSize;
    int minGridSize;
  	int realGridSize;

    int* mutex;
    hipMalloc((void**)&mutex, sizeof(int));
    hipMemset(mutex, 0, sizeof(int));

  	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, naive_multiexp_kernel_block_level_atomics, 0, 4 * N * 3 * WARP_SIZE);
  	realGridSize = (arr_len + blockSize - 1) / blockSize;

    ec_point point_at_infty = { 
        {0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000},
        {0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000001},
        {0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000}
    };

    hipMemcpy(out, &point_at_infty, sizeof(ec_point), hipMemcpyHostToDevice);

	std::cout << "Grid size: " << realGridSize << ",  min grid size: " << minGridSize << ",  blockSize: " << blockSize << std::endl;
	naive_multiexp_kernel_block_level_atomics<<<realGridSize, blockSize>>>(point_arr, power_arr, out, arr_len, mutex);

    hipFree(mutex);
}

//3) using block level reduction and recursion
//---------------------------------------------------------------------------------------------------------------------------------------------------------------

__global__ void naive_multiexp_kernel_block_level_recursion(affine_point* point_arr, uint256_g* power_arr, ec_point* out_arr, size_t arr_len)
{
    ec_point acc = point_at_infty();
    
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < arr_len)
	{   
        ec_point x = ECC_EXP(point_arr[tid], power_arr[tid]);
        acc = ECC_ADD(acc, x);
        tid += blockDim.x * gridDim.x;
	}

    acc = blockReduceSum(acc);
    
    if (threadIdx.x == 0)
        out_arr[blockIdx.x] = acc;
}

__global__ void naive_kernel_block_level_reduction(ec_point* in_arr, ec_point* out_arr, size_t arr_len)
{
    ec_point acc = point_at_infty();
    
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < arr_len)
    {   
        acc = ECC_ADD(acc, in_arr[tid]);
        tid += blockDim.x * gridDim.x;
	}

    acc = blockReduceSum(acc);

    if (threadIdx.x == 0)
        out_arr[blockIdx.x] = acc;
}

void naive_multiexp_kernel_block_level_recursion_driver(affine_point* point_arr, uint256_g* power_arr, ec_point* out_arr, size_t arr_len)
{
    int blockSize;
  	int minGridSize;
  	int realGridSize;

  	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, naive_multiexp_kernel_block_level_recursion, 0, 4 * N * 3 * WARP_SIZE);
  	realGridSize = (arr_len + blockSize - 1) / blockSize;;
    realGridSize = min(realGridSize, DEFAUL_NUM_OF_THREADS_PER_BLOCK);
    
	std::cout << "Real grid size: " << realGridSize << ",  min grid size: " << minGridSize << ",  blockSize: " << blockSize << std::endl;
	naive_multiexp_kernel_block_level_recursion<<<realGridSize, blockSize>>>(point_arr, power_arr, out_arr, arr_len);
    hipDeviceSynchronize();
    naive_kernel_block_level_reduction<<<1, DEFAUL_NUM_OF_THREADS_PER_BLOCK>>>(out_arr, out_arr, realGridSize);
}

//Pippenger
//---------------------------------------------------------------------------------------------------------------------------------------------------------
//---------------------------------------------------------------------------------------------------------------------------------------------------------
//---------------------------------------------------------------------------------------------------------------------------------------------------------

//Pippenger final exponentiation

__global__ void Pippenger_final_exponentiation(ec_point* in_arr, ec_point* out_arr, size_t arr_len)
{
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < arr_len)
    {   
        ec_point pt = in_arr[tid];

        for (size_t j = 0; j < threadIdx.x; j++)
            pt = ECC_DOUBLE(pt);
        
        out_arr[tid] = pt;

        tid += blockDim.x * gridDim.x;
	}
}

__global__ void multiexp_Pippenger(affine_point* point_arr, uint256_g* power_arr, ec_point* out_arr, size_t arr_len, int* mutex_arr)
{
    ec_point acc = point_at_infty();
    
    size_t start = (arr_len / gridDim.x) * blockIdx.x;
    size_t end = (arr_len / gridDim.x) * (blockIdx.x + 1);

    for (size_t i = start; i < end; i++)
    {
        if (get_bit(power_arr[i], threadIdx.x))
            acc = ECC_MIXED_ADD(acc, point_arr[i]);
    }

    while (atomicCAS(mutex_arr + threadIdx.x, 0, 1) != 0);
    out_arr[threadIdx.x] = ECC_ADD(out_arr[threadIdx.x], acc);
    atomicExch(mutex_arr + threadIdx.x, 0);   
}

void Pippenger_driver(affine_point* point_arr, uint256_g* power_arr, ec_point* out_arr, size_t arr_len)
{
    int blockSize;
  	int minGridSize;
  	int realGridSize;

    size_t M = 256;
    int* mutex_arr;
    hipMalloc((void**)&mutex_arr, sizeof(int) * M);
    hipMemset(mutex_arr, 0, sizeof(int) * M);

  	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, multiexp_Pippenger, 0, 4 * N * 3 * WARP_SIZE);
  	realGridSize = (arr_len + blockSize - 1) / blockSize;;

    //but here we need an array of such elements!

    ec_point point_at_infty = { 
        {0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000},
        {0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000001},
        {0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000, 0x00000000}
    };

    for (size_t j = 0 ; j < 256; j++)
    {
        hipMemcpy(out_arr + j, &point_at_infty, sizeof(ec_point), hipMemcpyHostToDevice);
    }
    
	std::cout << "Real grid size: " << realGridSize << ",  min grid size: " << minGridSize << ",  blockSize: " << blockSize << std::endl;

	multiexp_Pippenger<<<realGridSize, 256>>>(point_arr, power_arr, out_arr, arr_len, mutex_arr);
    hipDeviceSynchronize();

    Pippenger_final_exponentiation<<<1, 256>>>(out_arr, out_arr, 256);
    hipDeviceSynchronize();

    naive_kernel_block_level_reduction<<<1, 256>>>(out_arr, out_arr, 256);

    hipFree(mutex_arr);
}
