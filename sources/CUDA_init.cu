#include "cuda_structs.h"

#include <stdio.h>

bool CUDA_init()
{
    //first find suitable Cuda device
	//TBD: or split between several CUDA devices if possible
	int device_count;
	hipError_t cudaStatus = hipGetDeviceCount(&device_count);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipGetDeviceCount failed!");
		return false;
	}
	if (device_count == 0)
	{
		fprintf(stderr, "No suitable CUDA devices were found!");
		return false;
	}

	hipDeviceProp_t prop;
	cudaStatus = hipGetDeviceProperties(&prop, 0);

	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipGetDeviceCount failed!");
		return false;
	}

	//TODO: check if there are enough constant memory and other additional checks
	//set appropriate device
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return false;
	}

    return true;
}


